#include "hip/hip_runtime.h"
#include "srad.h"
#include <stdio.h>
#include <stdlib.h>

#define E_MAX   0.30f
#define P_TH   8 << 23
#define N_TH   -(8 << 23)

#define AB_DIV(a,b) a*(2.823f-1.882f*b)
#define X_DIV(x) (2.823f-1.882f*x)


//Additional Function
inline __device__ float mul(float a, float b)
{
	int m_sum = (__float_as_int(a) & 0x007FFFFF) + (__float_as_int(b) & 0x007FFFFF);
	int m_new = m_sum + 0x800000;

	if (m_sum & 0x800000) // M_a + M_b >= 1
	{
		m_new >>= 1;
		m_new += 0x800000;
	}

	m_new += (__float_as_int(a) & 0x7f800000) + (__float_as_int(b) & 0x7f800000) - 0x40000000; // exponent
	m_new |= (__float_as_int(a) & 0x80000000) ^ (__float_as_int(b) & 0x80000000); // sign

	return __int_as_float(m_new);
}


inline __device__ float sum(float a, float b)
{
	int d = (__float_as_int(a) & 0x7f800000) - (__float_as_int(b) & 0x7f800000);
	if (d > P_TH) return a; // abs(a) >>> abs(b)
	if (d < N_TH) return b; // abs(b) >>> abs(a)
	return a + b;
}


inline __device__ float sub(float a, float b)
{
	int d = (__float_as_int(a) & 0x7f800000) - (__float_as_int(b) & 0x7f800000);
	if (d > P_TH) return a; // abs(a) >>> abs(b)
	if (d < N_TH) return -b; // abs(b) >>> abs(a)
	return a - b;
}
//

__global__ void
srad_cuda_1(
	float *E_C,
	float *W_C,
	float *N_C,
	float *S_C,
	float * J_cuda,
	float * C_cuda,
	int cols,
	int rows,
	float q0sqr
)
{

	//block id
	int bx = blockIdx.x;
	int by = blockIdx.y;

	//thread id
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//indices
	int index = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
	int index_n = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + tx - cols;
	int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
	int index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty - 1;
	int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;

	float n, w, e, s, jc, g2, l, num, den, qsqr, c;

	//shared memory allocation
	__shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float temp_result[BLOCK_SIZE][BLOCK_SIZE];

	__shared__ float north[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float south[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float  east[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float  west[BLOCK_SIZE][BLOCK_SIZE];

	//load data to shared memory
	north[ty][tx] = J_cuda[index_n];
	south[ty][tx] = J_cuda[index_s];
	if (by == 0) {
		north[ty][tx] = J_cuda[BLOCK_SIZE * bx + tx];
	}
	else if (by == gridDim.y - 1) {
		south[ty][tx] = J_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * (BLOCK_SIZE - 1) + tx];
	}
	__syncthreads();

	west[ty][tx] = J_cuda[index_w];
	east[ty][tx] = J_cuda[index_e];

	if (bx == 0) {
		west[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + cols * ty];
	}
	else if (bx == gridDim.x - 1) {
		east[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * (gridDim.x - 1) + cols * ty + BLOCK_SIZE - 1];
	}

	__syncthreads();



	temp[ty][tx] = J_cuda[index];

	__syncthreads();

	jc = temp[ty][tx];

	if (ty == 0 && tx == 0) { //nw
		n = sub(north[ty][tx], jc);
		s = sub(temp[ty + 1][tx], jc);
		w = sub(west[ty][tx], jc);
		e = sub(temp[ty][tx + 1], jc);
	}
	else if (ty == 0 && tx == BLOCK_SIZE - 1) { //ne
		n = sub(north[ty][tx], jc);
		s = sub(temp[ty + 1][tx], jc);
		w = sub(temp[ty][tx - 1], jc);
		e = sub(east[ty][tx], jc);
	}
	else if (ty == BLOCK_SIZE - 1 && tx == BLOCK_SIZE - 1) { //se
		n = sub(temp[ty - 1][tx], jc);
		s = sub(south[ty][tx], jc);
		w = sub(temp[ty][tx - 1], jc);
		e = sub(east[ty][tx], jc);
	}
	else if (ty == BLOCK_SIZE - 1 && tx == 0) {//sw
		n = sub(temp[ty - 1][tx], jc);
		s = sub(south[ty][tx], jc);
		w = sub(west[ty][tx], jc);
		e = sub(temp[ty][tx + 1], jc);
	}

	else if (ty == 0) { //n
		n = sub(north[ty][tx], jc);
		s = sub(temp[ty + 1][tx], jc);
		w = sub(temp[ty][tx - 1], jc);
		e = sub(temp[ty][tx + 1], jc);
	}
	else if (tx == BLOCK_SIZE - 1) { //e
		n = sub(temp[ty - 1][tx], jc);
		s = sub(temp[ty + 1][tx], jc);
		w = sub(temp[ty][tx - 1], jc);
		e = sub(east[ty][tx], jc);
	}
	else if (ty == BLOCK_SIZE - 1) { //s
		n = sub(temp[ty - 1][tx], jc);
		s = sub(south[ty][tx], jc);
		w = sub(temp[ty][tx - 1], jc);
		e = sub(temp[ty][tx + 1], jc);
	}
	else if (tx == 0) { //w
		n = sub(temp[ty - 1][tx], jc);
		s = sub(temp[ty + 1][tx], jc);
		w = sub(west[ty][tx], jc);
		e = sub(temp[ty][tx + 1], jc);
	}
	else {  //the data elements which are not on the borders 
		n = sub(temp[ty - 1][tx], jc);
		s = sub(temp[ty + 1][tx], jc);
		w = sub(temp[ty][tx - 1], jc);
		e = sub(temp[ty][tx + 1], jc);
	}


	//g2 = (n * n + s * s + w * w + e * e) / (jc * jc);
	g2 = AB_DIV(sum(sum(mul(n, n), mul(s, s)), sum(mul(w, w), mul(e, e))), mul(jc, jc));
	//l = (n + s + w + e) / jc;
	l = AB_DIV(sum(sum(n, s), sum(w, e)), jc);

	num = (0.5*g2) - ((1.0 / 16.0)*(l*l));
	den = 1 + (.25*l);
	qsqr = AB_DIV(num, mul(den,den));

	// diffusion coefficent (equ 33)
	//den = (qsqr - q0sqr) / (q0sqr * (1 + q0sqr));
	den = AB_DIV(sub(qsqr, q0sqr), mul(q0sqr, 1 + q0sqr));

	//c = 1.0 / (1.0 + den);
	c = X_DIV(1.0 + den);
	
	// saturate diffusion coefficent
	if (c < 0) { temp_result[ty][tx] = 0; }
	else if (c > 1) { temp_result[ty][tx] = 1; }
	else { temp_result[ty][tx] = c; }

	__syncthreads();

	C_cuda[index] = temp_result[ty][tx];
	E_C[index] = e;
	W_C[index] = w;
	S_C[index] = s;
	N_C[index] = n;

}

__global__ void
srad_cuda_2(
	float *E_C,
	float *W_C,
	float *N_C,
	float *S_C,
	float * J_cuda,
	float * C_cuda,
	int cols,
	int rows,
	float lambda,
	float q0sqr
)
{
	//block id
	int bx = blockIdx.x;
	int by = blockIdx.y;

	//thread id
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//indices
	int index = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
	int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
	int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
	float cc, cn, cs, ce, cw, d_sum;

	//shared memory allocation
	__shared__ float south_c[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float  east_c[BLOCK_SIZE][BLOCK_SIZE];

	__shared__ float c_cuda_temp[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float c_cuda_result[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];

	//load data to shared memory
	temp[ty][tx] = J_cuda[index];

	__syncthreads();

	south_c[ty][tx] = C_cuda[index_s];

	if (by == gridDim.y - 1) {
		south_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * (BLOCK_SIZE - 1) + tx];
	}
	__syncthreads();


	east_c[ty][tx] = C_cuda[index_e];

	if (bx == gridDim.x - 1) {
		east_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * (gridDim.x - 1) + cols * ty + BLOCK_SIZE - 1];
	}

	__syncthreads();

	c_cuda_temp[ty][tx] = C_cuda[index];

	__syncthreads();

	cc = c_cuda_temp[ty][tx];

	if (ty == BLOCK_SIZE - 1 && tx == BLOCK_SIZE - 1) { //se
		cn = cc;
		cs = south_c[ty][tx];
		cw = cc;
		ce = east_c[ty][tx];
	}
	else if (tx == BLOCK_SIZE - 1) { //e
		cn = cc;
		cs = c_cuda_temp[ty + 1][tx];
		cw = cc;
		ce = east_c[ty][tx];
	}
	else if (ty == BLOCK_SIZE - 1) { //s
		cn = cc;
		cs = south_c[ty][tx];
		cw = cc;
		ce = c_cuda_temp[ty][tx + 1];
	}
	else { //the data elements which are not on the borders 
		cn = cc;
		cs = c_cuda_temp[ty + 1][tx];
		cw = cc;
		ce = c_cuda_temp[ty][tx + 1];
	}

	// divergence (equ 58)
	//d_sum = cn * N_C[index] + cs * S_C[index] + cw * W_C[index] + ce * E_C[index];
	d_sum = sum(sum(mul(cn, N_C[index]), mul(cs, S_C[index])), sum(mul(cw, W_C[index]), mul(ce, E_C[index])));

	// image update (equ 61)
	//c_cuda_result[ty][tx] = temp[ty][tx] + 0.25 * lambda * d_sum;
	c_cuda_result[ty][tx] = sum(temp[ty][tx], 0.25 * mul(lambda, d_sum));
	__syncthreads();

	J_cuda[index] = c_cuda_result[ty][tx];

}
